#include "hip/hip_runtime.h"
#include "cache.cuh"
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_fp16.h>
#include "../util.h"
#include "../util.cuh"

#define NUM_THREADS 512
#define NUM_BLOCKS 128

__global__ __launch_bounds__(NUM_THREADS)
void cache_rotate_kernel
(
    uint8_t* __restrict__ cache,
    const uint32_t* __restrict__ order,
    uint8_t* __restrict__ temp,
    size_t page_size,
    size_t rotate_len
)
{
    // Chunk for current CTA
    size_t block_size = CEIL_DIVIDE(page_size, gridDim.x);
    size_t block_beg = blockIdx.x * block_size;
    size_t block_end = MIN(block_beg + block_size, page_size);
    block_size = block_end - block_beg;
    if (!block_size) return;

    // Rotate pages
    auto copy = [&](uint8_t* dst, uint8_t* src)
    {
        for (int offset = threadIdx.x * 16; offset < block_size; offset += NUM_THREADS * 16)
            *((uint4*) (dst + offset)) = *((uint4*) (src + offset));
    };

    int i;
    copy(temp + block_beg, cache + page_size * (uint64_t) order[0] + block_beg);
    for (i = 0; i < rotate_len - 1; ++i)
        copy(cache + page_size * (uint64_t) order[i] + block_beg, cache + page_size * (uint64_t) order[i + 1] + block_beg);
    copy(cache + page_size * (uint64_t) order[i] + block_beg, temp + block_beg);
}

/*
Reorder cache pages
- cache, paged cache, shape (num_pages, ...), any dtype, contiguous
- order, sequence to rotate, shape (n,), dtype long
- temp, temp storage, sized as one cache page

Performs:

temp <- page[order[0]]
for a, b in pairwise(order):
    page[a] <- page[b]
page[order[-1]] <- temp
*/

void cache_rotate
(
    const at::Tensor& cache,
    const at::Tensor& order,
    const at::Tensor& temp
)
{
    const at::cuda::OptionalCUDAGuard device_guard(cache.device());
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

    TORCH_CHECK(cache.dim() > 1, "cache argument must have dim >= 2")
    TORCH_CHECK(order.dim() == 1, "order argument must have dim == 1")
    TORCH_CHECK_DTYPE(order, kInt);

    size_t num_pages = cache.size(0);
    size_t page_size = cache.nbytes() / num_pages;
    size_t rotate_len = order.size(0);

    TORCH_CHECK(temp.nbytes() == page_size, "temp tensor incorrect size");

    cache_rotate_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>
    (
        (uint8_t*) cache.data_ptr(),
        (const uint32_t*) order.data_ptr(),
        (uint8_t*) temp.data_ptr(),
        page_size,
        rotate_len
    );
}